#include "hip/hip_runtime.h"
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include <string>
#include <iostream>
#include <fstream>
#include "nqueens.h"

#define MAX_N 10
#define MAX_SOLUTIONS 724 // for N = 10

using namespace std;

int main() {
    bool PRINT_SOLUTIONS = false;
	CalculateAllSolutions(PRINT_SOLUTIONS);
}

void CalculateAllSolutions(bool print) {
    for (int N = 4; N <= MAX_N; N++) {
        vector<vector<int>> solutions;
        int solutionsCount;

        auto startTime = chrono::system_clock::now();
        CalculateSolutionsCUDA(N, solutions, &solutionsCount);
        auto endTime = chrono::system_clock::now();

        auto total = endTime - startTime;
        auto totalTime = chrono::duration_cast<chrono::microseconds>(total).count();
        solutionsCount = solutions.size();

        if (print)
            PrintSolutions(N, solutions);
        printf("N=%d, solutions=%d, run time=%lld\n", N, solutionsCount, totalTime);

    }
}

// sets up the GPU and executes the kernel
void CalculateSolutionsCUDA(int N, vector<vector<int>>& solutions, int* solutionsCount) {
    __int64 possibleCombinations = powl(N, N); // use powl and __int64 to fit the biggest numbers

    size_t solutionsSize = sizeof(int[MAX_N]) * MAX_SOLUTIONS; // a solutions is an array of size N so calculate the memory for int[MAX_N]

    // initialise host memory
    *solutionsCount = 0;
    int* solutionsRaw = (int*)malloc(solutionsSize);

    // initialise device memory
    int* solutionsBuffer = nullptr;
    int* countBuffer = nullptr;

    // allocate device memory
    hipMalloc((void**)&solutionsBuffer, solutionsSize);
    hipMalloc((void**)&countBuffer, sizeof(int));

    // copy the starting number of solutions to device to initialise the in-device counter 
    hipMemcpy(countBuffer, solutionsCount, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 1024;
    // if there are less possible combinations than blockSize, have only one block of blockSize
    __int64 gridSize = (possibleCombinations / blockSize < 1) ? 1 : possibleCombinations / blockSize + 1;

    // call the kernel
    GenerateCombinations <<<gridSize, blockSize >>> (N, possibleCombinations, solutionsBuffer, countBuffer);

    // copy the results back to the host and free device memory 
    hipMemcpy(solutionsRaw, solutionsBuffer, solutionsSize, hipMemcpyDeviceToHost);
    hipMemcpy(solutionsCount, countBuffer, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(countBuffer);
    hipFree(solutionsBuffer);

    // process the solutions array to the final solutions vector
    for (int i = 0; i < *solutionsCount; i++) {
        std::vector<int> solution(solutionsRaw + (N * i), solutionsRaw + (N * (i + 1)));
        solutions.push_back(solution);
    }

    free(solutionsRaw); // clean up host
}

// Main CUDA kernel for generating solutions
__global__ void GenerateCombinations(int N, __int64 possibleCombinations, int* solutionsBuffer, int* countBuffer) {
    __int64 currentCombination = threadIdx.x + blockIdx.x * blockDim.x; // this is also the conversion base

    // check if the kernel has not gone over the possible combination number 
    // this is possible because the kernel may execute on a bigger number of threads 
    // than there are possible combinations
    if (currentCombination >= possibleCombinations)
        return;

    int rowIndices[MAX_N];
    if (GenerateValidCombination(N, currentCombination, &rowIndices[0])) {
        int solutionIndex = atomicAdd(countBuffer, 1); // this returns the value of countBuffer before incrementing so will act as the index to solutionsBuffer
        for (int column = 0; column < N; column++) // store combination in solutions buffer
            solutionsBuffer[N * solutionIndex + column] = rowIndices[column];
    }
}

__device__ bool GenerateValidCombination(int N, __int64 currentCombination, int* rowIndices) {
    for (int column = 0; column < N; column++) {
        rowIndices[column] = currentCombination % N;
        currentCombination /= N;

        if (!CheckIfValidSolution(column, rowIndices))
            return false;
    }
    return true;
}

// adapted to reflect column placement instead of row
__device__ bool CheckIfValidSolution(int lastFilledColumn, int* rowIndices)
{
    // Check against other queens
    for (int column = 0; column < lastFilledColumn; ++column)
    {
        // check the rows
        if (rowIndices[column] == rowIndices[lastFilledColumn])
            return false;
        // check the 2 diagonals
        const auto col1 = rowIndices[lastFilledColumn] - (lastFilledColumn - column);
        const auto col2 = rowIndices[lastFilledColumn] + (lastFilledColumn - column);
        if (rowIndices[column] == col1 || rowIndices[column] == col2)
            return false;
    }
    return true;
}

/// <summary>
/// Print all solutions for N to the console
/// Modified from the given one to rpint column-by-column instead of row-by-row
/// </summary>
/// <param name="N"></param>
/// <param name="solutions"></param>
void PrintSolutions(int N, vector<vector<int>>& solutions) {
    std::string text;
    text.resize(N * (N + 1) + 1);
    text.back() = '\n'; // add extra line at the end
    for (const auto& solution : solutions)
    {
        // go through each column
        for (int column = 0; column <= N; column++)
        {
            if (column != N) {
                int rowIndex = solution[column];
                for (int row = 0; row < N; row++)
                    text[row * (N + 1) + column] = rowIndex == row ? 'X' : '.';
            }
            // if last column, add the endlines 
            else {
                for (int row = 0; row < N; row++)
                    text[row * (N + 1) + column] = '\n';
            }
        }
        cout << text << "\n";
    }
}