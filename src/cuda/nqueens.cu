#include "hip/hip_runtime.h"
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include <string>
#include <iostream>
#include <fstream>
#include "nqueens.h"

#define MAX_N 10

using namespace std;

int main() {
    bool PRINT_SOLUTIONS = false;
	CalculateAllSolutions(PRINT_SOLUTIONS);
}

void CalculateAllSolutions(bool print) {
    ofstream data("data.csv");
    for (int N = 4; N <= MAX_N; N++) {
        data << "N " << N << "\n";
        double meanTime = 0;
        vector<vector<int>> solutions;
        int solutionsCount = 0;

        auto startTime = chrono::system_clock::now();
        CalculateSolutionsCUDA(N, solutions, &solutionsCount);
        auto endTime = chrono::system_clock::now();

        auto total = endTime - startTime;
        auto totalTime = chrono::duration_cast<chrono::microseconds>(total).count();
        data << totalTime << "\n";
        meanTime += totalTime;
        solutionsCount = solutions.size();
        printf("N=%d, solutions=%d, run time=%lld\n", N, solutionsCount, totalTime);

        if (print)
            PrintSolutions(N, solutions);
    }
}

void CalculateSolutionsCUDA(int N, vector<vector<int>>& solutions, int* solutionsCount) {
    *solutionsCount = 0;
    int* solutionsBuffer = nullptr;
    int* countBuffer = nullptr;

    __int64 possibleCombinations = powl(N, N);

    size_t solutionsSize = powl(N, 5) * sizeof(int*); 
    hipMalloc((void**)&solutionsBuffer, solutionsSize);
    hipMalloc((void**)&countBuffer, sizeof(int));

    hipMemcpy(countBuffer, solutionsCount, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 1024;
    // if there are less possible combinations than blockSize, have only one block of blockSize
    long long int gridSize = (possibleCombinations / blockSize < 1) ? 1 : possibleCombinations / blockSize + 1;

    GenerateValidCombination <<<gridSize, blockSize >>> (N, possibleCombinations, solutionsBuffer, countBuffer);

    hipDeviceSynchronize();

    hipMemcpy(solutionsCount, countBuffer, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(countBuffer);

    int* solutionsRaw = (int*)malloc(solutionsSize);
    hipMemcpy(solutionsRaw, solutionsBuffer, solutionsSize, hipMemcpyDeviceToHost);
    hipFree(solutionsBuffer);

    for (__int64 i = 0; i < *solutionsCount; i++) {
        std::vector<int> solution;
        for (int j = 0; j < N; j++)
            solution.push_back(solutionsRaw[N * i + j]);
        solutions.push_back(solution);
    }

    free(solutionsRaw);

}

__global__ void GenerateValidCombination(int N, __int64 possibleCombinations, int* solutionsBuffer, int* countBuffer) {
    __int64 currentCombination = threadIdx.x + blockIdx.x * blockDim.x; // this is also the conversion base

    // check if the kernel has not gone over the possible combination number 
    // this is possible because the kernel may execute on a bigger number of threads 
    // than there are possible combinations
    if (currentCombination >= possibleCombinations)
        return;

    int rowIndices[MAX_N];
    if (GenerateCombination(N, currentCombination, &rowIndices[0])) {

        int solutionIndex = atomicAdd(countBuffer, 1); // this returns the value of countBuffer before incrementing so will act as the index to solutionsBuffer
        for (int column = 0; column < N; column++)
            solutionsBuffer[N * solutionIndex + column] = rowIndices[column];
    }
}

__device__ bool GenerateCombination(int N, __int64 currentCombination, int* rowIndices) {
    for (int column = 0; column < N; column++) {
        rowIndices[column] = currentCombination % N;
        currentCombination /= N;

        if (!CheckIfValidSolution(column, rowIndices))
            return false;
    }
    return true;
}

// adapted to reflect column placement instead of row
__device__ bool CheckIfValidSolution(int lastFilledColumn, int* rowIndices)
{
    // Check against other queens
    for (int column = 0; column < lastFilledColumn; ++column)
    {
        if (rowIndices[column] == rowIndices[lastFilledColumn])
            return false;
        // check the 2 diagonals
        const auto col1 = rowIndices[lastFilledColumn] - (lastFilledColumn - column);
        const auto col2 = rowIndices[lastFilledColumn] + (lastFilledColumn - column);
        if (rowIndices[column] == col1 || rowIndices[column] == col2)
            return false;
    }
    return true;
}

/// <summary>
/// Print all solutions for N to the console
/// Modified from the given one to rpint column-by-column instead of row-by-row
/// </summary>
/// <param name="N"></param>
/// <param name="solutions"></param>
void PrintSolutions(int N, vector<vector<int>>& solutions) {
    std::string text;
    text.resize(N * (N + 1) + 1);
    text.back() = '\n'; // add extra line at the end
    for (const auto& solution : solutions)
    {
        // go through each column
        for (int column = 0; column <= N; column++)
        {
            if (column != N) {
                int rowIndex = solution[column];
                for (int row = 0; row < N; row++)
                    text[row * (N + 1) + column] = rowIndex == row ? 'X' : '.';
            }
            // if last column, add the endlines 
            else {
                for (int row = 0; row < N; row++)
                    text[row * (N + 1) + column] = '\n';
            }
        }
        cout << text << "\n";
    }
}

